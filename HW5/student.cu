#include "hip/hip_runtime.h"
/* Udacity HW5
   Histogramming for Speed

   The goal of this assignment is compute a histogram
   as fast as possible.  We have simplified the problem as much as
   possible to allow you to focus solely on the histogramming algorithm.

   The input values that you need to histogram are already the exact
   bins that need to be updated.  This is unlike in HW3 where you needed
   to compute the range of the data and then do:
   bin = (val - valMin) / valRange to determine the bin.

   Here the bin is just:
   bin = val

   so the serial histogram calculation looks like:
   for (i = 0; i < numElems; ++i)
     histo[val[i]]++;

   That's it!  Your job is to make it run as fast as possible!

   The values are normally distributed - you may take
   advantage of this fact in your implementation.

*/


#include "utils.h"
#include "reference_calc.cpp"

#define BLOCK_SIZE_MAX 512				// i.e. maximum number of threads per block
#define GRID_SIZE_MAX 512				// i.e. maximum number of blocks
#define NUMBER_OF_ELEMS_PER_THREAD 8	// Number of elements (values) to be processed per thread

// #define HISTO2

__global__
void consolidateKernel(const unsigned int* const d_In,	//INPUT: values
               unsigned int* const d_Out,				//OUPUT: histogram
			   int numVals,
			   unsigned int valsOffset,
			   unsigned int numBins)
{
	int threadsPerBlock = blockDim.x * blockDim.y;

    int blockId = blockIdx.x + (blockIdx.y * gridDim.x);

    int threadId = threadIdx.x + (threadIdx.y * blockDim.x);

    int myId = (blockId * threadsPerBlock) + threadId;

	if ( ((valsOffset + myId) < numVals) &&
		 (myId < numBins) )
	{
		d_Out[myId] +=
			d_In[myId];
	}
}

__global__
void histogramKernel(const unsigned int* const d_In,	//INPUT: values
               unsigned int* const d_Out,				//OUPUT: histogram
               int numVals,
			   unsigned int valsOffset,
			   unsigned int numBins)
{
	int threadsPerBlock = blockDim.x * blockDim.y;

	int threadsPerGrid = threadsPerBlock * gridDim.x * gridDim.y;

    int blockId = blockIdx.x + (blockIdx.y * gridDim.x);

    int threadId = threadIdx.x + (threadIdx.y * blockDim.x);

    int myId = (blockId * threadsPerBlock) + threadId;

	for (
		int _step = 0;
		_step < NUMBER_OF_ELEMS_PER_THREAD;
		_step++)
	{
		int _myTrueId =
			myId + _step * threadsPerGrid;

		if ( (_myTrueId + valsOffset) >= numVals )
		{
			break;
		}
		else
		{
			unsigned int _in =
				d_In[_myTrueId];

			atomicAdd(&(d_Out[_in]), 1);
		}
	}
}

#if defined(HISTO2)

__global__
void histogramKernel2(const unsigned int* const d_In, //INPUT: values
               unsigned int* const d_Out,      //OUPUT: histogram
               int numVals,
			   unsigned int valsOffset,
			   unsigned int numBins)
{
	extern __shared__ unsigned int s_histogramKernel_Out[];

	int threadsPerBlock = blockDim.x * blockDim.y;

	int threadsPerGrid = threadsPerBlock * gridDim.x * gridDim.y;

    int blockId = blockIdx.x + (blockIdx.y * gridDim.x);

    int threadId = threadIdx.x + (threadIdx.y * blockDim.x);

	for (
		int i = 0;
		i < (numBins / threadsPerBlock);
		i++)
	{
		int _index =
			i * threadsPerBlock + threadId;

		if (_index < numBins)
		{
			s_histogramKernel_Out[_index] =
				d_Out[_index];
		}
	}

	__syncthreads();

    int myId = (blockId * threadsPerBlock) + threadId;

	for (
		int _step = 0;
		_step < NUMBER_OF_ELEMS_PER_THREAD;
		_step++)
	{
		int _myTrueId =
			myId + _step * threadsPerGrid;

		if ( (_myTrueId + valsOffset) >= numVals )
		{
			break;
		}
		else
		{
			unsigned int _in =
				d_In[_myTrueId];

			atomicAdd(&(s_histogramKernel_Out[_in]), 1);
		}
	}

	__syncthreads();

	for (
		int i = 0;
		i < (numBins / threadsPerBlock);
		i++)
	{
		int _index =
			i * threadsPerBlock + threadId;

		if (_index < numBins)
		{
			d_Out[_index] =
				s_histogramKernel_Out[_index];
		}
	}
}

#define PARALLEL_HISTOS 2

#endif

void computeHistogram(const unsigned int* const d_In, //INPUT: values
                      unsigned int* const d_Out,      //OUTPUT: histogram
                      const unsigned int numBins,
                      const unsigned int numElems)
{
	unsigned int _numElemsProcessed = 0;

	dim3 _block(BLOCK_SIZE_MAX);

#if defined(HISTO2)

	unsigned int* d_OutTmp = 0;

	checkCudaErrors(
		hipMalloc(
			&d_OutTmp,
			numBins * sizeof(unsigned int) * PARALLEL_HISTOS));

	if (d_OutTmp)
	{
		checkCudaErrors(
			hipMemset(
				d_OutTmp,
				0x0,
				numBins * sizeof(unsigned int) * PARALLEL_HISTOS));
	}

	bool ping = true;

#endif

	while (_numElemsProcessed < numElems)
	{
		int numElemGroupsLeft =
			(numElems - _numElemsProcessed - 1) / NUMBER_OF_ELEMS_PER_THREAD + 1;

		int _gridSize =
			(numElemGroupsLeft - 1) / BLOCK_SIZE_MAX + 1;

		_gridSize =
			_gridSize < GRID_SIZE_MAX ?
				_gridSize :
				GRID_SIZE_MAX;

		dim3 _grid(_gridSize);

#if !defined(HISTO2)

		// Launch a kernel on the GPU with one thread for each element.
		histogramKernel<<<_grid, _block, (numBins * sizeof(unsigned int))>>>
			(&d_In[_numElemsProcessed],
			 d_Out,
			 numElems,
			 _numElemsProcessed,
			 numBins);

#else
		unsigned int *_d_OutTmp =
			ping ?
				&d_OutTmp[0]:
				&d_OutTmp[(PARALLEL_HISTOS >> 1) * numBins];

		// Launch a kernel on the GPU with one thread for each element.
		histogramKernel<<<_grid, _block, (numBins * sizeof(unsigned int))>>>
			(&d_In[_numElemsProcessed],
			 _d_OutTmp,
			 numElems,
			 _numElemsProcessed,
			 numBins);

		dim3 _blockConsolidate(BLOCK_SIZE_MAX);

		int _gridSizeConsolidate =
			(numBins - 1) / BLOCK_SIZE_MAX + 1;

		dim3 _gridConsolidate(_gridSizeConsolidate);

		consolidateKernel<<<_gridConsolidate, _blockConsolidate>>>
			(_d_OutTmp,
			d_Out,
			numElems,
			_numElemsProcessed,
			numBins);

		checkCudaErrors(
			hipMemset(
				_d_OutTmp,
				0x0,
				numBins * sizeof(unsigned int)));

		ping = !ping;
#endif

		_numElemsProcessed +=
			_gridSize * BLOCK_SIZE_MAX * NUMBER_OF_ELEMS_PER_THREAD;
	}

  //if you want to use/launch more than one kernel,
  //feel free
  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

#if defined(HISTO2)

  if (d_OutTmp)
  {
	hipFree(d_OutTmp);
  }

#endif

    /*
  delete[] h_vals;
  delete[] h_histo;
  delete[] your_histo;*/
}
