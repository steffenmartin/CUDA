#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Definition Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/


#include "reference_calc.cpp"
#include "utils.h"
#include "hip/hip_runtime.h"
#include <math.h>

// #define USE_PRINTF_FOR_DEBUG

// #define ENABLE_REF_CHECK

#define BLOCK_SIZE_MAX_X 16						// i.e. maximum number of threads per block (x dimension)
												// Note: In this particular application it should be a power
												//       of 2
#define BLOCK_SIZE_MAX_Y 16						// i.e. maximum number of threads per block (y dimension)
												// Note: In this particular application it should be a power
												//       of 2

#define BLOCK_SIZE_HISTO_MAX_X 22				// i.e. maximum number of threads per block (x dimension)
#define BLOCK_SIZE_HISTO_MAX_Y 22				// i.e. maximum number of threads per block (y dimension)

#define BLOCK_SIZE_SCAN_MAX_X 512				// i.e. maximum number of threads per block (x dimension)
#define BLOCK_SIZE_SCAN_MAX_Y 1					// i.e. maximum number of threads per block (y dimension)

__global__
	void global_find_min(float *d_Out,
						 float *d_OutMax,
						 float *d_In,
						 float *d_InMax,
						 int numRows, int numCols)
{
	// Determine position in 2D (for boundary checking)
	const int2 thread_2D_pos =
		make_int2( blockIdx.x * blockDim.x + threadIdx.x,
				   blockIdx.y * blockDim.y + threadIdx.y);

	if ( thread_2D_pos.x >= numCols ||
		 thread_2D_pos.y >= numRows )
	{
		return;
	}
	else
	{
		// Determine position in 1D (walking along columns and jumping to the next row at the end of the colum).
		const int _thread_1D_pos =
			thread_2D_pos.y * numCols + thread_2D_pos.x;

		// Let's calculate total number of pixels (just once)
		const int numPixelTotal =
			numRows * numCols;

		// Let's determine the number of pixel this block is working on
		const int numPixelBlock =
			(blockDim.x * blockDim.y);

		int thread_1D_pos =
			blockIdx.x * numPixelBlock + threadIdx.y * blockDim.x + threadIdx.x;
		thread_1D_pos +=
			blockIdx.y * blockDim.y * numCols;

		// Let's determine the index inside of this block
		int tid =
			threadIdx.y * blockDim.x + threadIdx.x;

		// do reduction in global mem
		for (unsigned int s = numPixelBlock / 2; s > 0; s >>= 1)
		{
			if (tid < s &&
				(thread_1D_pos + s) < numPixelTotal)
			{
				d_In[thread_1D_pos] =
					min(d_In[thread_1D_pos], d_In[thread_1D_pos + s]);
				d_InMax[thread_1D_pos] =
					max(d_InMax[thread_1D_pos], d_InMax[thread_1D_pos + s]);
			}
			__syncthreads();        // make sure all min/max at one stage are done!
		}

		// only thread 0 writes result for this block back to global mem
		if (tid == 0)
		{
			d_Out[thread_1D_pos / numPixelBlock] = d_In[thread_1D_pos];
			d_OutMax[thread_1D_pos / numPixelBlock] = d_InMax[thread_1D_pos];
		}
	}
}

__global__
	void simple_histo(unsigned int *d_bins,
					  const float *d_In,
					  const unsigned int BIN_COUNT,
					  float _min,
					  float _range,
					  int numRows, int numCols)
{
	// Determine position in 2D (for boundary checking)
	const int2 thread_2D_pos =
		make_int2( blockIdx.x * blockDim.x + threadIdx.x,
				   blockIdx.y * blockDim.y + threadIdx.y);

	if ( thread_2D_pos.x >= numCols ||
		 thread_2D_pos.y >= numRows )
	{
		return;
	}
	else
	{
		// const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;
		const int thread_1D_pos =
			blockIdx.y * blockDim.y * gridDim.x +
			blockIdx.x * blockDim.x * blockDim.y +
			threadIdx.y * blockDim.x +
			threadIdx.x;

		float myItem = d_In[thread_1D_pos];
		
		// int myBin = ((myItem - d_Min[0]) / (d_Max[0] - d_Min[0])) * BIN_COUNT;

		unsigned int myBin = 
			min(
				static_cast<unsigned int>(BIN_COUNT - 1),
				static_cast<unsigned int>((myItem - _min) / _range * BIN_COUNT));

		atomicAdd(&(d_bins[myBin]), 1);
	}
}

__global__
	void scan_naive_exclusive(unsigned int *d_In,
							  unsigned int *d_Out,
							  int numElements)
{
	int myId = threadIdx.x + blockDim.x * blockIdx.x;

	if (myId < numElements)
	{
		for (int i = 0; i < myId; i++)
		{
			d_Out[myId] +=
				d_In[i];
		}
	}
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
#if defined(USE_PRINTF_FOR_DEBUG)

	printf("Image is %i columns x %i rows\n",
		numCols,
		numRows);

	printf("Number of bins is %i\n",
		numBins);

#endif

  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
	   */

	int gridSizeX = (numCols - 1) / BLOCK_SIZE_MAX_X + 1;
	int gridSizeY = (numRows - 1) / BLOCK_SIZE_MAX_Y + 1;

	// Block size (i.e., number of threads per block)
	dim3 blockSize(BLOCK_SIZE_MAX_X, BLOCK_SIZE_MAX_Y, 1);

	// Compute grid size (i.e., number of blocks per kernel launch)
	// from the image size and and block size.
	dim3 gridSize(gridSizeX, gridSizeY, 1);

	float *d_IntermediateIn;
	float *d_IntermediateInMax;
	// Allocate memory on the device for storing the intermediate input values and copy them
	checkCudaErrors(
		hipMalloc(
			&d_IntermediateIn,
			sizeof(float) * numRows * numCols));
	checkCudaErrors(
		hipMalloc(
			&d_IntermediateInMax,
			sizeof(float) * numRows * numCols));
	checkCudaErrors(
		hipMemcpy(
			d_IntermediateIn,
			d_logLuminance,
			sizeof(float) * numRows * numCols,
			hipMemcpyDeviceToDevice));
	checkCudaErrors(
		hipMemcpy(
			d_IntermediateInMax,
			d_logLuminance,
			sizeof(float) * numRows * numCols,
			hipMemcpyDeviceToDevice));

	float *d_IntermediateOut;
	float *d_IntermediateOutMax;
	// Allocate memory on the device for storing the intermediate output values
	checkCudaErrors(
		hipMalloc(
			&d_IntermediateOut,
			sizeof(float) * gridSizeX * gridSizeY));
	checkCudaErrors(
		hipMalloc(
			&d_IntermediateOutMax,
			sizeof(float) * gridSizeX * gridSizeY));
	checkCudaErrors(
		hipMemset(
			d_IntermediateOut,
			0x0,
			sizeof(float) * gridSizeX * gridSizeY));
	checkCudaErrors(
		hipMemset(
			d_IntermediateOutMax,
			0x0,
			sizeof(float) * gridSizeX * gridSizeY));

	float *d_MinOut;
	float *d_MaxOut;
	float *d_MinMaxOut;
	// Allocate memory on the device for storing the output value
	checkCudaErrors(
		hipMalloc(
			&d_MinOut,
			sizeof(float)));
	checkCudaErrors(
		hipMalloc(
			&d_MaxOut,
			sizeof(float)));
	checkCudaErrors(
		hipMalloc(
			&d_MinMaxOut,
			2 * sizeof(float)));
	checkCudaErrors(
		hipMemset(
			d_MinOut,
			0x0,
			sizeof(float)));
	checkCudaErrors(
		hipMemset(
			d_MaxOut,
			0x0,
			sizeof(float)));
	checkCudaErrors(
		hipMemset(
			d_MinMaxOut,
			0x0,
			2 * sizeof(float)));

#if defined(USE_PRINTF_FOR_DEBUG)

	float *h_Intermediate =
		// new float[sizeof(float) * gridSizeX * gridSizeY];
		new float[sizeof(float) * numRows * numCols];
	memset(
		h_Intermediate,
		0x0,
		// sizeof(float) * gridSizeX * gridSizeY);
		sizeof(float) * numRows * numCols);

	checkCudaErrors(
		hipMemcpy(
			h_Intermediate,
			d_logLuminance,
			// sizeof(float) * gridSizeX * gridSizeY,
			sizeof(float) * numRows * numCols,
			hipMemcpyDeviceToHost));

	checkCudaErrors(
		hipMemcpy(
			h_Intermediate,
			d_IntermediateIn,
			// sizeof(float) * gridSizeX * gridSizeY,
			sizeof(float) * numRows * numCols,
			hipMemcpyDeviceToHost));

	float h_Out = 0;

	printf("Blocksize\tX: %i\tY: %i\tZ: %i\n",
		blockSize.x,
		blockSize.y,
		blockSize.z);

	printf("Gridsize\tX: %i\tY: %i\tZ: %i\n",
		gridSize.x,
		gridSize.y,
		gridSize.z);

#endif

	global_find_min<<<gridSize, blockSize>>>
		(d_IntermediateOut,
		 d_IntermediateOutMax,
		 d_IntermediateIn,
		 d_IntermediateInMax,
		 numRows,
		 numCols);

#if defined(USE_PRINTF_FOR_DEBUG)

	checkCudaErrors(
		hipMemcpy(
			h_Intermediate,
			d_IntermediateOut,
			sizeof(float) * gridSizeX * gridSizeY,
			hipMemcpyDeviceToHost));

	checkCudaErrors(
		hipMemcpy(
			h_Intermediate,
			d_IntermediateOutMax,
			sizeof(float) * gridSizeX * gridSizeY,
			hipMemcpyDeviceToHost));

#endif

	global_find_min<<<1, blockSize>>>
		(&d_MinMaxOut[0],
		 &d_MinMaxOut[1],
		 d_IntermediateOut,
		 d_IntermediateOutMax,
		 gridSizeX,
		 gridSizeY);

#if defined(USE_PRINTF_FOR_DEBUG)

	checkCudaErrors(
		hipMemcpy(
			&h_Out,
			&d_MinMaxOut[0],
			sizeof(float),
			hipMemcpyDeviceToHost));

	printf("Min: %f\n", h_Out);

	checkCudaErrors(
		hipMemcpy(
			&h_Out,
			&d_MinMaxOut[1],
			sizeof(float),
			hipMemcpyDeviceToHost));

	printf("Max: %f\n", h_Out);

#endif

	/*
    2) subtract them to find the range
	*/

	float h_MinMaxOut[2];

	checkCudaErrors(
		hipMemcpy(
			&h_MinMaxOut[0],
			d_MinMaxOut,
			2 * sizeof(float),
			hipMemcpyDeviceToHost));

	min_logLum =
		h_MinMaxOut[0];
	max_logLum =
		h_MinMaxOut[1];
	float _logLumRange = max_logLum - min_logLum;

	/*
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
	   */

	unsigned int *d_Bins;

	// Allocate memory on the device for storing the intermediate output values
	checkCudaErrors(
		hipMalloc(
			&d_Bins,
			sizeof(unsigned int) * numBins));
	checkCudaErrors(
		hipMemset(
			d_Bins,
			0x0,
			sizeof(unsigned int) * numBins));

#if defined(USE_PRINTF_FOR_DEBUG)

	unsigned int *h_Bins =
		new unsigned int[numBins];

	memset(h_Bins, 0x0, sizeof(unsigned int) * numBins);

#endif

	gridSizeX = (numCols - 1) / BLOCK_SIZE_HISTO_MAX_X + 1;
	gridSizeY = (numRows - 1) / BLOCK_SIZE_HISTO_MAX_Y + 1;

	// Block size (i.e., number of threads per block)
	blockSize.x = BLOCK_SIZE_HISTO_MAX_X;
	blockSize.y = BLOCK_SIZE_HISTO_MAX_Y;

	// Compute grid size (i.e., number of blocks per kernel launch)
	// from the image size and and block size.
	gridSize.x = gridSizeX;
	gridSize.y = gridSizeY;

	simple_histo<<<gridSize, blockSize>>>(
		d_Bins,
		d_logLuminance,
		numBins,
		h_MinMaxOut[0],
		_logLumRange,
		numRows,
		numCols);

#if defined(USE_PRINTF_FOR_DEBUG)

	checkCudaErrors(
		hipMemcpy(
			h_Bins,
			d_Bins,
			sizeof(unsigned int) * numBins,
			hipMemcpyDeviceToHost));
#endif

	/*
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */

	gridSizeX = (numBins - 1) / BLOCK_SIZE_SCAN_MAX_X + 1;
	gridSizeY = 1;

	// Block size (i.e., number of threads per block)
	blockSize.x = BLOCK_SIZE_SCAN_MAX_X;
	blockSize.y = BLOCK_SIZE_SCAN_MAX_Y;

	// Compute grid size (i.e., number of blocks per kernel launch)
	// from the image size and and block size.
	gridSize.x = gridSizeX;
	gridSize.y = gridSizeY;

	scan_naive_exclusive<<<gridSize, blockSize>>>(
		d_Bins,
		d_cdf,
		numBins);

  /****************************************************************************
  * You can use the code below to help with debugging, but make sure to       *
  * comment it out again before submitting your assignment for grading,       *
  * otherwise this code will take too much time and make it seem like your    *
  * GPU implementation isn't fast enough.                                     *
  *                                                                           *
  * This code generates a reference cdf on the host by running the            *
  * reference calculation we have given you.  It then copies your GPU         *
  * generated cdf back to the host and calls a function that compares the     *
  * the two and will output the first location they differ.                   *
  * ************************************************************************* */

#if defined(ENABLE_REF_CHECK)

  float *h_logLuminance = new float[numRows * numCols];
  unsigned int *h_cdf   = new unsigned int[numBins];
  unsigned int *h_your_cdf = new unsigned int[numBins];
  checkCudaErrors(hipMemcpy(h_logLuminance, d_logLuminance, numCols * numRows * sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(h_your_cdf, d_cdf, numBins * sizeof(unsigned int), hipMemcpyDeviceToHost));

  referenceCalculation(h_logLuminance, h_cdf, numRows, numCols, numBins);

  //compare the results of the CDF
  // checkResultsExact(h_cdf, h_your_cdf, numBins);
  checkResultsEps(h_cdf, h_your_cdf, numBins, 3, 10);
 
  delete[] h_logLuminance;
  delete[] h_cdf; 
  delete[] h_your_cdf;

#endif

	checkCudaErrors(hipFree(d_IntermediateIn));
	checkCudaErrors(hipFree(d_IntermediateInMax));
	checkCudaErrors(hipFree(d_IntermediateOut));
	checkCudaErrors(hipFree(d_IntermediateOutMax));
	checkCudaErrors(hipFree(d_Bins));

#if defined(USE_PRINTF_FOR_DEBUG)

	delete []h_Intermediate;
	delete []h_Bins;

#endif

}
