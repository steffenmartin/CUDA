#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include "timer.h"

#define MAX_THREAD_BLOCK_SIZE 512

// #define ADD
// #define SCAN
#define SCAN_EXCLUSIVE

#if defined(ADD)

hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

#endif

#if defined(SCAN)

hipError_t scanWithCuda(int *c, const int *a, size_t size);

// Inclusive Scan (Hillis/Steele)

__global__ void scanKernelInclusive(int *c, const int *a, size_t size, size_t offset)
{
    int myId = 
		threadIdx.x;

	if (((myId - offset) < size) &&
		(myId >= offset))
	{
		c[myId] = a[myId];

		__syncthreads();

		size_t _stepsLeft =
			size;

		unsigned int _neighbor =
			1;

		while (_stepsLeft)
		{
			int op1 = c[myId];
			int op2 = 0;

			if ((myId - offset) >= _neighbor)
			{
				op2 =
					c[myId - _neighbor];
			}
			else
			{
				break;
			}

			__syncthreads();

			c[myId] =
				op1 + op2;

			__syncthreads();

			_stepsLeft >>= 1;
			_neighbor <<= 1;
		}

		if (offset > 0)
		{
			c[myId] +=
				c[offset - 1];
		}
	}
}

#endif

#if defined(SCAN_EXCLUSIVE)

hipError_t scanExclusiveWithCuda(const unsigned int *h_In, unsigned int *h_Out, size_t size);

// Exclusive Scan (Blelloch)

__global__
	void scanKernelExclusive(const unsigned int *d_In,
							 unsigned int *d_Out,
							 size_t size,
							 size_t offset,
							 bool isLastCall)
{
	// Stores boundary values to account for sizes that are not powers of 2
	__shared__ unsigned int _boundaryValueCurrent;
	__shared__ unsigned int _finalAdd;
	unsigned int _finalRemember;

	int myId = 
		threadIdx.x;

	if (myId == 0)
	{
		_boundaryValueCurrent = 0;

		_finalRemember =
			d_In[offset + size - 1];

		if (offset > 0)
		{
			_finalAdd =
				d_Out[0] + d_Out[offset - 1];
		}
	}

	__syncthreads();

	if (myId < size)
	{
		// Initial data fetch
		d_Out[myId + offset] =
			d_In[myId + offset];

		__syncthreads();

		// Used to track how many steps are left by right-shifting its value
		// (i.e. implicitely calculating log2 of the size)
		size_t _stepsLeft =
			size;

		// Which neighbor to the left has to be added?
		unsigned int _neighbor =
			1;

		// Is it my turn to add?
		unsigned int _selfMask =
			1;

		// Step 1: Adding neighbors

		while (_stepsLeft)
		{
			if ((_selfMask & myId) == _selfMask)
			{
				d_Out[myId + offset] +=
					d_Out[(myId + offset) - _neighbor];
			}

			_stepsLeft >>= 1;
			_neighbor <<= 1;
			_selfMask <<= 1;
			_selfMask++;

			__syncthreads();
		}

		// Step 2: Down-sweep and adding neighbors again

		// Adjustment to properly start
		_selfMask--;
		_selfMask >>= 1;
		_neighbor >>= 1;
		_stepsLeft = size;

		while (_stepsLeft)
		{
			bool _fillInBoundaryValue =
				true;

			if ((_selfMask & myId) == _selfMask)
			{
				unsigned int _tmp =
					d_Out[myId + offset];

				d_Out[myId + offset] +=
					d_Out[(myId + offset) - _neighbor];

				d_Out[(myId + offset) - _neighbor] =
					_tmp;

				_fillInBoundaryValue =
					false;
			}

			__syncthreads();

			// Cross-sweep of boundary value

			unsigned int _selfMaskCrossSweep =
				_selfMask >> 1;

			if (_fillInBoundaryValue)
			{
				if (((_selfMask & myId) ^ _selfMaskCrossSweep) == 0)
				{
					if ((myId + _neighbor) >= size)
					{
						unsigned int _boundaryValueTmp =
							_boundaryValueCurrent + d_Out[(myId + offset)];

						d_Out[myId + offset] =
							_boundaryValueCurrent;

						_boundaryValueCurrent =
							_boundaryValueTmp;
					}
				}
			}
			
			_selfMask--;
			_selfMask >>= 1;
			_neighbor >>= 1;
			_stepsLeft >>= 1;

			__syncthreads();
		}

		if (offset > 0)
		{
			d_Out[(myId + offset)] +=
				_finalAdd;
		}

		__syncthreads();
	}

	if (myId == 0)
	{
		if (isLastCall)
		{
			d_Out[0] =
				0;
		}
		else
		{
			d_Out[0] =
				_finalRemember;
		}
	}
}

#endif

int main()
{
    const int arraySize = 13;
    const unsigned int a[arraySize] = { 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13 };
    const unsigned int b[arraySize] = { 10, 20, 30, 40, 50, 60, 70, 80, 90, 100, 110, 120, 130 };
    unsigned int c[arraySize] = { 0 };

	hipError_t cudaStatus =
		hipSuccess;

	float _elapsed = 0;

#if defined(ADD)

    // Add vectors in parallel.
    cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
        c[0], c[1], c[2], c[3], c[4]);

#endif

#if defined(SCAN)

	// Timing
	{
		GpuTimer timer;
		timer.Start();
		// Scan vector in parallel.
		cudaStatus = scanWithCuda(c, a, arraySize);
		timer.Stop();
		_elapsed = timer.Elapsed();
	}
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "scanWithCuda failed!");
        return 1;
    }

	// hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }
	else
	{
		printf("%f msecs.\n\n", _elapsed);

		for (int i = 0; i < arraySize; i++)
		{
			printf(
				"element %i -> in: %d\t out: %d\n",
				i,
				a[i],
				c[i]);
		}
	}

#endif

#if defined(SCAN_EXCLUSIVE)

	// Timing
	{
		GpuTimer timer;
		timer.Start();
		// Scan vector in parallel.
		cudaStatus = scanExclusiveWithCuda(a, c, arraySize);
		timer.Stop();
		_elapsed = timer.Elapsed();
	}
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "scanExclusiveWithCuda failed!");
        return 1;
    }

	// hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    }
	else
	{
		printf("%f msecs.\n\n", _elapsed);

		for (int i = 0; i < arraySize; i++)
		{
			printf(
				"element %i -> in: %d\t out: %d\n",
				i,
				a[i],
				c[i]);
		}
	}

#endif

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

	getchar();

    return 0;
}

#if defined(ADD)

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, size_t size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}

#endif

#if defined(SCAN)

hipError_t scanWithCuda(int *c, const int *a, size_t size)
{
	int *dev_a = 0;
	int *dev_c = 0;

    hipError_t cudaStatus =
		size < MAX_THREAD_BLOCK_SIZE ?
			hipSuccess :
			hipErrorUnknown;

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Size cannot exceed maximum thread block size of %d", MAX_THREAD_BLOCK_SIZE);
        goto Error;
	}

	// Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for two vectors (one input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	// Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	 // Launch a kernel on the GPU with one thread for each element.
	scanKernelInclusive<<<1, size>>>(dev_c, dev_a, size >> 1, 0);
	scanKernelInclusive<<<1, size>>>(dev_c, dev_a, size >> 1 - 1, size >> 1);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);

	return cudaStatus;
}

#endif

#if defined(SCAN_EXCLUSIVE)

hipError_t scanExclusiveWithCuda(const unsigned int *h_In, unsigned int *h_Out, size_t size)
{
	unsigned int *d_In = 0;
	unsigned int *d_Out = 0;

    hipError_t cudaStatus =
		size < MAX_THREAD_BLOCK_SIZE ?
			hipSuccess :
			hipErrorUnknown;

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Size cannot exceed maximum thread block size of %d", MAX_THREAD_BLOCK_SIZE);
        goto Error;
	}

	// Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for two vectors (one input, one output)    .
    cudaStatus = hipMalloc((void**)&d_Out, size * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&d_In, size * sizeof(unsigned int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	// Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(d_In, h_In, size * sizeof(unsigned int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	 // Launch a kernel on the GPU with one thread for each element.
	scanKernelExclusive<<<1, size>>>(d_In, d_Out, size >> 1, 0, false);
	scanKernelExclusive<<<1, size>>>(d_In, d_Out, size - (size >> 1), size >> 1, true);

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(h_Out, d_Out, size * sizeof(unsigned int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(d_Out);
    hipFree(d_In);

	return cudaStatus;
}

#endif